#include "hip/hip_runtime.h"
#include <GL/glew.h>
#include "Dependencies\freeglut\freeglut.h"

#include <stdio.h>
#include <stdlib.h>

#include "pgm.h"
#include "hip/hip_runtime.h"
#include ""
#include <cuda_gl_interop.h>

#include <stdio.h>

// Interop with a pixel buffer 
// Runs both sobel and blur
// Testing keep map and fusion

int run = 1;

GLuint tex;
GLuint buf;

GLuint out_tex;
GLuint out_buf;

GLuint in_tex;
GLuint in_buf;

hipArray *out_array;
hipArray *in_array;

hipArray *out_temp;
char *out_iqm;
hipArray *in_temp;
char *in_iqm;

char* keepMap;


GLuint width;
GLuint height;


hipGraphicsResource *out_resource;
hipGraphicsResource *in_resource;

hipGraphicsResource *resource;


// Reference the textures in CUDA kernels
texture<char, 2, hipReadModeElementType> out_texRef;
texture<char, 2, hipReadModeElementType> in_texRef;


// Used for checking errors
void checkError(int line) {
	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s) at %d !\n", hipGetErrorString(err), line);
		system("PAUSE");
		exit(EXIT_FAILURE);
	}
}


// Fuses the the two images using regular data accessing
__global__ void fuse2(char *keep, char *in, char *out, char* data) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;

	if (keep[x] == 1) {
		data[x] = in[x];
	}
	else {
		data[x] = out[x];
	}
}

// Fuses the two images using texture references
__global__ void fuse(char *keep, char* data, unsigned int pitch) {
	unsigned char *p =
		(unsigned char *)(((char *)data) + blockIdx.x*pitch);

	for (int i = threadIdx.x; i < pitch; i += blockDim.x) {
		if (keep[i] == 1) {
			p[i] = tex2D(in_texRef, (float)i, (float)blockIdx.x);
		}
		else {
			p[i] = tex2D(out_texRef, (float)i, (float)blockIdx.x);
		}
	}
}

// Generates the keep map using regular accessing
__global__ void keep2(char* keep, char *in, char *out, unsigned int pitch) {

	for (int i = threadIdx.x; i < pitch; i += blockDim.x) {
		if (in[i] > out[i]) {
			keep[i] = 1;
		}
		else {
			keep[i] = 0;
		}
	}
	
}

// Genereates the keep map using texture references
__global__ void keep(char *out, int w, int h, unsigned int pitch) {

	unsigned char *p =
		(unsigned char *)(((char *)out) + blockIdx.x*pitch);

	for (int i = threadIdx.x; i < w; i += blockDim.x) {
		if (tex2D(in_texRef, (float)i, (float)blockIdx.x) > tex2D(out_texRef, (float)i, (float)blockIdx.x)) {
			p[i] = 1;
		}
		else {
			p[i] = 0;
		}
	}

}

// Sobel operator kernel for the output texture reference
__global__ void out_sobel(char *original, int w, int h, unsigned int pitch) {


	unsigned char *p =
		(unsigned char *)(((char *)original) + blockIdx.x*pitch);

	for (int i = threadIdx.x; i < w; i += blockDim.x)
	{
		unsigned char ul = tex2D(out_texRef, (float)i - 1, (float)blockIdx.x - 1);
		unsigned char um = tex2D(out_texRef, (float)i + 0, (float)blockIdx.x - 1);
		unsigned char ur = tex2D(out_texRef, (float)i + 1, (float)blockIdx.x - 1);
		unsigned char ml = tex2D(out_texRef, (float)i - 1, (float)blockIdx.x + 0);
		unsigned char mm = tex2D(out_texRef, (float)i + 0, (float)blockIdx.x + 0);
		unsigned char mr = tex2D(out_texRef, (float)i + 1, (float)blockIdx.x + 0);
		unsigned char ll = tex2D(out_texRef, (float)i - 1, (float)blockIdx.x + 1);
		unsigned char lm = tex2D(out_texRef, (float)i + 0, (float)blockIdx.x + 1);
		unsigned char lr = tex2D(out_texRef, (float)i + 1, (float)blockIdx.x + 1);

		short Horz = ur + 2 * mr + lr - ul - 2 * ml - ll;
		short Vert = ul + 2 * um + ur - ll - 2 * lm - lr;
		short Sum = (short)((abs((int)Horz) + abs((int)Vert)));

		if (Sum < 0)
		{
			p[i] = 0;
		}
		else if (Sum > 0xff)
		{
			p[i] = 0xff;
		}
		else {
			p[i] = (unsigned char)Sum;
		}
	}

}

// Sobel operator kernel for the input texture reference
__global__ void in_sobel(char *original, int w, int h, unsigned int pitch) {


	unsigned char *p =
		(unsigned char *)(((char *)original) + blockIdx.x*pitch);

	for (int i = threadIdx.x; i < w; i += blockDim.x)
	{
		unsigned char ul = tex2D(in_texRef, (float)i - 1, (float)blockIdx.x - 1);
		unsigned char um = tex2D(in_texRef, (float)i + 0, (float)blockIdx.x - 1);
		unsigned char ur = tex2D(in_texRef, (float)i + 1, (float)blockIdx.x - 1);
		unsigned char ml = tex2D(in_texRef, (float)i - 1, (float)blockIdx.x + 0);
		unsigned char mm = tex2D(in_texRef, (float)i + 0, (float)blockIdx.x + 0);
		unsigned char mr = tex2D(in_texRef, (float)i + 1, (float)blockIdx.x + 0);
		unsigned char ll = tex2D(in_texRef, (float)i - 1, (float)blockIdx.x + 1);
		unsigned char lm = tex2D(in_texRef, (float)i + 0, (float)blockIdx.x + 1);
		unsigned char lr = tex2D(in_texRef, (float)i + 1, (float)blockIdx.x + 1);

		short Horz = ur + 2 * mr + lr - ul - 2 * ml - ll;
		short Vert = ul + 2 * um + ur - ll - 2 * lm - lr;
		short Sum = (short)((abs((int)Horz) + abs((int)Vert)));

		if (Sum < 0)
		{
			p[i] = 0;
		}
		else if (Sum > 0xff)
		{
			p[i] = 0xff;
		}
		else {
			p[i] = (unsigned char)Sum;
		}
	}

}

// NEEDS UPDATING, OVERFLOW BUG
// Blur kernel for the output texture reference
__global__ void out_blur(char *original, int w, int h, unsigned int pitch) {


	char *p = (((char *)original) + blockIdx.x*pitch);

	for (int i = threadIdx.x; i < w; i += blockDim.x)
	{
		float ul = ((float)tex2D(out_texRef, (float)i - 1, (float)blockIdx.x - 1)) * (float)(1.f / 9);
		float um = ((float)tex2D(out_texRef, (float)i + 0, (float)blockIdx.x - 1)) * (float)(1.f / 9);
		float ur = ((float)tex2D(out_texRef, (float)i + 1, (float)blockIdx.x - 1)) * (float)(1.f / 9);
		float ml = ((float)tex2D(out_texRef, (float)i - 1, (float)blockIdx.x + 0)) * (float)(1.f / 9);
		float mm = ((float)tex2D(out_texRef, (float)i + 0, (float)blockIdx.x + 0)) * (float)(1.f / 9);
		float mr = ((float)tex2D(out_texRef, (float)i + 1, (float)blockIdx.x + 0)) * (float)(1.f / 9);
		float ll = ((float)tex2D(out_texRef, (float)i - 1, (float)blockIdx.x + 1)) * (float)(1.f / 9);
		float lm = ((float)tex2D(out_texRef, (float)i + 0, (float)blockIdx.x + 1)) * (float)(1.f / 9);
		float lr = ((float)tex2D(out_texRef, (float)i + 1, (float)blockIdx.x + 1)) * (float)(1.f / 9);

		p[i] = (char)(ul + um + ur + ml + mm + mr + ll + lm + lr);
	}

}

// NEEDS UPDATING, OVERFLOW BUG
// Blur kernel for the input texture reference
__global__ void in_blur(char *original, int w, int h, unsigned int pitch) {


	char *p = (((char *)original) + blockIdx.x*pitch);

	for (int i = threadIdx.x; i < w; i += blockDim.x)
	{
		float ul = ((float)tex2D(in_texRef, (float)i - 1, (float)blockIdx.x - 1)) * (float)(1.f / 9);
		float um = ((float)tex2D(in_texRef, (float)i + 0, (float)blockIdx.x - 1)) * (float)(1.f / 9);
		float ur = ((float)tex2D(in_texRef, (float)i + 1, (float)blockIdx.x - 1)) * (float)(1.f / 9);
		float ml = ((float)tex2D(in_texRef, (float)i - 1, (float)blockIdx.x + 0)) * (float)(1.f / 9);
		float mm = ((float)tex2D(in_texRef, (float)i + 0, (float)blockIdx.x + 0)) * (float)(1.f / 9);
		float mr = ((float)tex2D(in_texRef, (float)i + 1, (float)blockIdx.x + 0)) * (float)(1.f / 9);
		float ll = ((float)tex2D(in_texRef, (float)i - 1, (float)blockIdx.x + 1)) * (float)(1.f / 9);
		float lm = ((float)tex2D(in_texRef, (float)i + 0, (float)blockIdx.x + 1)) * (float)(1.f / 9);
		float lr = ((float)tex2D(in_texRef, (float)i + 1, (float)blockIdx.x + 1)) * (float)(1.f / 9);

		p[i] = (char)(ul + um + ur + ml + mm + mr + ll + lm + lr);
	}

}



void initData() {
	_PGMData pic;
	//readPGM("lena512.pgm", &pic);
	readPGM("pgm/000.pgm", &pic);

	// Update width and height variables
	width = pic.col;
	height = pic.row;
	printf("%d %d \n", width, height);

	// Create channel description for cuda arrays
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<char>();

	// Setup cuda arrays for temp and output
	hipMallocArray(&out_array, &channelDesc, pic.col, pic.row);
	hipMemcpyToArray(out_array, 0, 0, pic.matrix, pic.col * pic.row * sizeof(char), hipMemcpyHostToDevice);

	hipMallocArray(&out_temp, &channelDesc, pic.col, pic.row);
	hipMemcpyToArray(out_temp, 0, 0, pic.matrix, pic.col * pic.row * sizeof(char), hipMemcpyHostToDevice);

	
	// Might need new reference to update?
	// _PGMData pic0;

	//readPGM("lena512.pgm", &pic);
	readPGM("pgm/001.pgm", &pic);

	// Setup cuda arrays for temp and input
	hipMallocArray(&in_array, &channelDesc, pic.col, pic.row);
	hipMemcpyToArray(in_array, 0, 0, pic.matrix, pic.col * pic.row * sizeof(char), hipMemcpyHostToDevice);

	hipMallocArray(&in_temp, &channelDesc, pic.col, pic.row);
	hipMemcpyToArray(in_temp, 0, 0, pic.matrix, pic.col * pic.row * sizeof(char), hipMemcpyHostToDevice);

	
	// Allocate space for other buffers
	hipMalloc((void **)&keepMap, width * height * sizeof(char));
	hipMalloc((void **)&out_iqm, width * height * sizeof(char));
	hipMalloc((void **)&in_iqm, width * height * sizeof(char));
}

void initTexture() {

	// Initialize buffer pixel unpack buffer
	glGenBuffers(1, &buf);
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER, buf);
																										// GLBufferSubData for adding different data
	glBufferData(GL_PIXEL_UNPACK_BUFFER, height * width * sizeof(char), NULL, GL_STREAM_DRAW);			// This prevents unneccesary copy
	hipGraphicsGLRegisterBuffer(&resource, buf, cudaGraphicsMapFlagsWriteDiscard);		// Registers resoure as device pointer

	// Initialize texture
	glGenTextures(1, &tex);
	glBindTexture(GL_TEXTURE_2D, tex);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_EDGE);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_EDGE);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
	glTexImage2D(GL_TEXTURE_2D, 0, GL_LUMINANCE8, width, height, 0, GL_RED, GL_UNSIGNED_BYTE, NULL);
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);
	glBindTexture(GL_TEXTURE_2D, 0);
}





void display() {

	// Map data pointer to the final cuda resource for display
	char *data = NULL;
	size_t bytes;

	hipGraphicsMapResources(1, &resource, 0);
	hipGraphicsResourceGetMappedPointer((void **)&data, &bytes, resource);
	
	// Bind cuda arrays to texture references
	hipBindTextureToArray(out_texRef, out_array);
	hipBindTextureToArray(in_texRef, in_array);

	// Run sobel kernels for output and input and copy data to buffers
	// Extra copy shouldn't be necessary when old images will be thrown away
	out_sobel << <height, 16 >> > (out_iqm, width, height, width);
	hipMemcpyToArray(out_temp, 0, 0, out_iqm, height * width * sizeof(char), hipMemcpyDeviceToDevice);
	in_sobel << <height, 16 >> > (in_iqm, width, height, width);
	hipMemcpyToArray(in_temp, 0, 0, in_iqm, height * width * sizeof(char), hipMemcpyDeviceToDevice);

	// Unbind texture references and bind to new buffers
	// Again shouldn't be necessary in the future
	hipUnbindTexture(out_texRef);
	hipUnbindTexture(in_texRef);
	hipBindTextureToArray(out_texRef, out_temp);
	hipBindTextureToArray(in_texRef, in_temp);

	// Run blur kernels and write to first buffer
	out_blur << <height, 16 >> > (out_iqm, width, height, width);
	in_blur << <height, 16 >> > (in_iqm, width, height, width);

	// Run keep map kernel to generate keep map
	keep2 <<< height, 16 >>> (keepMap, in_iqm, out_iqm, width);

	// Unbind texture references and bind to first buffers
	// Again shouldn't be necessary in the future
	hipUnbindTexture(out_texRef);
	hipUnbindTexture(in_texRef);
	hipBindTextureToArray(out_texRef, out_array);
	hipBindTextureToArray(in_texRef, in_array);

	// Run fuse kernel and write to the mapped data buffer
	fuse << < height, 16 >> > (keepMap, data, width);

	// Unbind texture references
	hipUnbindTexture(out_texRef);
	hipUnbindTexture(in_texRef);
	
	// Unmap resources
	hipGraphicsUnmapResources(1, &resource, 0); // THIS NEEDS TO BE DONE TO USE IT WITH OPENGL

	// Go through OpenGL drawing to quads
	// Can update this at some point to used modern OpenGL and shaders
	glClear(GL_COLOR_BUFFER_BIT);

	glBindTexture(GL_TEXTURE_2D, tex);
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER, buf);
	glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, width, height, GL_RED, GL_UNSIGNED_BYTE, NULL);
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);

	glEnable(GL_TEXTURE_2D);

	glBegin(GL_QUADS);
	glTexCoord2f(0.0, 1.0); glVertex3f(-1.0, -1.0, 0.5);
	glTexCoord2f(1.0, 1.0); glVertex3f(1.0, -1.0, 0.5);
	glTexCoord2f(1.0, 0.0); glVertex3f(1.0, 1.0, 0.5);
	glTexCoord2f(0.0, 0.0); glVertex3f(-1.0, 1.0, 0.5);
	glEnd();

	glBindTexture(GL_TEXTURE_2D, 0);

	glDisable(GL_TEXTURE_2D);

	glutSwapBuffers();
	glutPostRedisplay(); // Creates continuous loop
}

int main(int argc, char **argv) {

	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_DEPTH | GLUT_DOUBLE | GLUT_RGBA);
	// Add required version for openGL if necessary
	glutInitWindowPosition(600, 100);
	glutInitWindowSize(1024, 800);	// Should set this to be width and height, possibly with constants
	glutCreateWindow("OpenGL First Window");

	glewInit();

	// Initializations
	initData();
	initTexture();

	// OpenGL setup
	glutDisplayFunc(display);
	glutMainLoop();

	//getchar();
	return 0;
}